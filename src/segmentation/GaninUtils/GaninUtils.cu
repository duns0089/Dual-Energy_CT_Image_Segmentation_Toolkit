#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This application demonstrates an approach to the image segmentation
 * trees construction. It is based on Boruvka's MST algorithm.
 * Here's the complete list of references:
 * 1) V. Vineet et al, "Fast Minimum Spanning Tree for
 *    Large Graphs on the GPU";
 * 2) P. Felzenszwalb et al, "Efficient Graph-Based Image Segmentation";
 * 3) A. Ion et al, "Considerations Regarding the Minimum Spanning
 *    Tree Pyramid Segmentation Method".
 */

// System includes.
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// STL includes.
#include <iostream>
#include <fstream>
#include <iterator>
#include <vector>
#include <list>
#include <deque>
#include <algorithm>

// Thrust library includes.
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/for_each.h>
#include <thrust/reduce.h>
#include <thrust/unique.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/adjacent_difference.h>
#include <thrust/find.h>

#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

// Sample framework includes.
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <Helpers.h>

// Project includes.
// #include "common.cuh"

// Kernels.
#include "kernels.cuh"

using std::cin;
using std::cout;
using std::deque;
using std::endl;
using std::list;
using std::vector;

inline void WaitEnter()
{
    std::cout << "Press Enter to continue...";
    while (std::cin.get() != '\n')
    {
        // waiting
    }
}

// Very simple von Neumann middle-square prng.  rand() is different across
// various OS platforms, which makes testing and the output inconsistent.
int myrand(void)
{
    static int seed = 72191;
    char sq[22];

    seed *= seed;
    sprintf(sq, "%010d", seed);
    // pull the middle 5 digits out of sq
    sq[8] = 0;
    seed = atoi(&sq[3]);

    return seed;
}

// Simple memory pool class. It is nothing more than array of fixed-sized
// arrays.
template <typename T>
class DeviceMemoryPool
{
private:
    uint chunkSize_, chunkRawSize_;
    thrust::device_ptr<void> basePtr_;

    list<thrust::device_ptr<T>> chunks_;

public:
    // The parameters of the constructor are as follows:
    // 1) uint chunkSize --- size of the particular array;
    // 2) uint chunksCount --- number of fixed-sized arrays.
    DeviceMemoryPool(uint chunkSize, uint chunksCount) : chunkSize_(chunkSize)
    {
        // cout << "here" << endl;
        chunkRawSize_ = (chunkSize * sizeof(T) + 511) & ~511;

        try
        {
            basePtr_ = thrust::device_malloc(chunkRawSize_ * chunksCount);
        }
        catch (thrust::system_error &e)
        {
            cout << "Pool memory allocation failed (1) (" << e.what() << ")" << endl;
            exit(EXIT_FAILURE);
        }
        catch (thrust::system::detail::bad_alloc &e)
        {
            cout << "Pool memory allocation failed (2) (" << e.what() << ")" << endl;
            exit(EXIT_FAILURE);
        }

        for (uint chunkIndex = 0; chunkIndex < chunksCount; ++chunkIndex)
        {
            chunks_.push_back(thrust::device_ptr<T>(reinterpret_cast<T *>(static_cast<char *>(basePtr_.get()) + chunkRawSize_ * chunkIndex)));
        }
    }

    ~DeviceMemoryPool()
    {
        try
        {
            thrust::device_free(basePtr_);
        }
        catch (thrust::system_error &e)
        {
            cout << "Pool memory allocation failed (" << e.what() << ")"
                 << endl;
            exit(EXIT_FAILURE);
        }
    }

    // Returns an address of the first available array
    // in the memory pool.
    thrust::device_ptr<T> get()
    {
        thrust::device_ptr<T> ptr(chunks_.back());
        chunks_.pop_back();

        return ptr;
    }

    // Pushes an address stored in "ptr" to the list
    // of available arrays of the memory pool.
    // It should be noted that it is user who is responsible for returning
    // the previously requested memory to the appropriate pool.
    inline void put(const thrust::device_ptr<T> &ptr)
    {
        chunks_.push_back(ptr);
    }

    uint totalFreeChunks() const
    {
        return chunks_.size();
    }
};

// Graph structure.
struct Graph
{
    Graph() {}

    Graph(uint verticesCount, uint edgesCount) : vertices(verticesCount),
                                                 edges(edgesCount),
                                                 weights(edgesCount)
    {
    }

    // This vector stores offsets for each vertex in "edges" and "weights"
    // vectors. For example:
    // "vertices[0]" is an index of the first outgoing edge of vertex #0,
    // "vertices[1]" is an index of the first outgoing edge of vertex #1, etc.
    vector<uint> vertices;

    // This vector stores indices of endpoints of the corresponding edges.
    // For example, "edges[vertices[0]]" is the first neighbouring vertex
    // of vertex #0.
    vector<uint> edges;

    // This vector stores weights of the corresponding edges.
    vector<float> weights;
};

// Simple segmentation tree class.
// Each level of the tree corresponds to the segmentation.
// See "Level" class for the details.
class Pyramid
{
public:
    void addLevel(uint totalSuperNodes, uint totalNodes, thrust::device_ptr<uint> superVerticesOffsets, thrust::device_ptr<uint> verticesIDs)
    {
        levels_.push_back(Level(totalSuperNodes, totalNodes));
        levels_.back().buildFromDeviceData(superVerticesOffsets, verticesIDs);
    }

    uint levelsCount() const
    {
        return static_cast<uint>(levels_.size());
    }

    void dump(int3 num_image_voxels_xyz) const
    {

        uint total_x, total_y, total_z;
        total_x = num_image_voxels_xyz.x;
        total_y = num_image_voxels_xyz.y;
        total_z = num_image_voxels_xyz.z;

        uint levelIndex = 0;

        uint requiredDigitsCount = static_cast<uint>(log10(static_cast<float>(levelsCount()))) + 1;

        for (LevelsIterator level = levels_.rbegin(); level != levels_.rend(); ++level, ++levelIndex)
        {
            std::string filename = "GaninbMST_lvl" + std::to_string(levelIndex) + "_" + std::to_string(total_x) + "x" + std::to_string(total_y) + "x" + std::to_string(total_z) + "_U32Bit.raw";

            dumpLevel(level, num_image_voxels_xyz, filename);
        }
    }

private:
    // Level of the segmentation tree.
    class Level
    {
    public:
        Level(uint totalSuperNodes, uint totalNodes) : superNodesOffsets_(totalSuperNodes), nodes_(totalNodes)
        {
        }

        void buildFromDeviceData(
            thrust::device_ptr<uint> superVerticesOffsets,
            thrust::device_ptr<uint> verticesIDs)
        {
            checkCudaErrors(
                hipMemcpy(&(superNodesOffsets_[0]),
                           superVerticesOffsets.get(),
                           sizeof(uint) * superNodesOffsets_.size(),
                           hipMemcpyDeviceToHost));

            checkCudaErrors(
                hipMemcpy(&(nodes_[0]),
                           verticesIDs.get(),
                           sizeof(uint) * nodes_.size(),
                           hipMemcpyDeviceToHost));
        }

    private:
        friend class Pyramid;

        // The pair of the following vectors describes the
        // relation between the consecutive levels.
        // Consider an example. Let the index of the current level be n.
        // Then nodes of level #(n-1) with indices stored in
        // "nodes[superNodesOffsets_[0]]",
        // "nodes[superNodesOffsets_[0] + 1]",
        // ...,
        // "nodes[superNodesOffsets_[1] - 1]"
        // correspond to vertex #0 of level #n. An so on.
        vector<uint> superNodesOffsets_;
        vector<uint> nodes_;
    };

    typedef list<Level>::const_reverse_iterator LevelsIterator;

    // Dumps level to the file "level_n.ppm" where n
    // is index of the level. Segments are drawn in random colors.
    // void dumpLevel(LevelsIterator level, uint width, uint height, const char *filename) const
    void dumpLevel(LevelsIterator level, int3 num_image_voxels_xyz, std::string filename) const
    {

        uint total_x, total_y, total_z;
        total_x = num_image_voxels_xyz.x;
        total_y = num_image_voxels_xyz.y;
        total_z = num_image_voxels_xyz.z;

        deque<std::pair<uint, uint>> nodesQueue;

        uint totalSegments;

        {
            const vector<uint> &superNodesOffsets = level->superNodesOffsets_;
            const vector<uint> &nodes = level->nodes_;

            totalSegments = static_cast<uint>(superNodesOffsets.size());

            for (uint superNodeIndex = 0, nodeIndex = 0; superNodeIndex < superNodesOffsets.size(); ++superNodeIndex)
            {

                uint superNodeEnd = superNodeIndex + 1 < superNodesOffsets.size() ? superNodesOffsets[superNodeIndex + 1] : static_cast<uint>(nodes.size());

                for (; nodeIndex < superNodeEnd; ++nodeIndex)
                {
                    nodesQueue.push_back(std::make_pair(nodes[nodeIndex], superNodeIndex));
                }
            }
        }

        ++level;

        while (level != levels_.rend())
        {
            uint superNodesCount = static_cast<uint>(nodesQueue.size());

            const vector<uint> &superNodesOffsets = level->superNodesOffsets_;
            const vector<uint> &nodes = level->nodes_;

            while (superNodesCount--)
            {
                std::pair<uint, uint> currentNode = nodesQueue.front();
                nodesQueue.pop_front();

                uint superNodeBegin = superNodesOffsets[currentNode.first];

                uint superNodeEnd =
                    currentNode.first + 1 < superNodesOffsets.size() ? superNodesOffsets[currentNode.first + 1] : static_cast<uint>(nodes.size());

                for (uint nodeIndex = superNodeBegin; nodeIndex < superNodeEnd; ++nodeIndex)
                {
                    nodesQueue.push_back(std::make_pair(nodes[nodeIndex], currentNode.second));
                }
            }

            ++level;
        }

        vector<uint> colors(totalSegments);

        for (uint colorIndex = 0; colorIndex < totalSegments; ++colorIndex)
        {
            colors[colorIndex] = colorIndex + 1;
        }
        cout << "Total segments: " << totalSegments << endl;

        uint32_t *image = new uint32_t[total_x * total_y * total_z];

        while (!nodesQueue.empty())
        {

            std::pair<uint, uint> currentNode = nodesQueue.front();
            nodesQueue.pop_front();

            uint pixelIndex = currentNode.first;
            uint pixelSegment = currentNode.second;

            image[pixelIndex] = colors[pixelSegment];
        }

        saveSegmentedImage(image, num_image_voxels_xyz, filename);

        delete[] image;
    }

    void saveSegmentedImage(uint32_t *image, int3 num_image_voxels_xyz, std::string file_name) const
    {
        int x, y, z;
        x = num_image_voxels_xyz.x;
        y = num_image_voxels_xyz.y;
        z = num_image_voxels_xyz.z;

        // Define the specific file to open based on the view
        std::ostringstream filename;
        filename << output_path << "/" << file_name;

        std::string file_open;
        file_open = filename.str();

        // Open File and ensure it can be opened
        std::ofstream file(file_open, std::ios::binary);
        if (!file.is_open())
        {
            std::cout << "ERROR: Can't create or access " << file_open << "." << std::endl;
        }

        // Read data into buffer
        for (int length = 0; length < y; ++length)
        {
            for (int width = 0; width < x; ++width)
            {
                for (int height = 0; height < z; ++height)
                {
                    int index = INDEX3D(height, width, length, z, x);
                    file.write(reinterpret_cast<char *>(&image[index]), sizeof(uint32_t));
                }
            }
        }
        file.close();
        cout << "saved " << file_name << endl;
        return;
    }

    std::string output_path = "output";

    list<Level> levels_;
};

// The class that encapsulates the main algorithm.
class SegmentationTreeBuilder
{
public:
    SegmentationTreeBuilder() : verticesCount_(0), edgesCount_(0) {}

    ~SegmentationTreeBuilder() {}

    // Repeatedly invokes the step of the algorithm
    // until the limiting segmentation is found.
    // Returns time (in ms) spent on building the tree.
    float run(const Graph &graph, Pyramid &segmentations, int3 num_image_voxels_xyz)
    {
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);

        // Allocate required memory pools. We need just 4 types of arrays.
        MemoryPoolsCollection pools = {
            DeviceMemoryPool<uint>(static_cast<uint>(graph.vertices.size()), kUintVerticesPoolsRequired),
            DeviceMemoryPool<float>(static_cast<uint>(graph.vertices.size()), kFloatVerticesPoolsRequired),
            DeviceMemoryPool<uint>(static_cast<uint>(graph.edges.size()), kUintEdgesPoolsRequired),
            DeviceMemoryPool<float>(static_cast<uint>(graph.edges.size()), kFloatEdgesPoolsRequired)};

        // Initialize internal variables
        try
        {
            initalizeData(graph, pools);
        }
        catch (thrust::system_error &e)
        {
            cout << "Initialization failed (" << e.what() << ")" << endl;
            exit(EXIT_FAILURE);
        }

        // Run steps
        AlgorithmStatus status;

        try
        {
            do
            {
                status = invokeStep(pools, segmentations, num_image_voxels_xyz);

            } while (status != ALGORITHM_FINISHED);
        }
        catch (thrust::system_error &e)
        {
            cout << "Algorithm failed (" << e.what() << ")" << endl;
            exit(EXIT_FAILURE);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);

        return elapsedTime;
    }

private:
    void printMemoryUsage()
    {
        size_t availableMemory, totalMemory, usedMemory;

        hipMemGetInfo(&availableMemory, &totalMemory);
        usedMemory = totalMemory - availableMemory;

        cout << "Device memory: used " << usedMemory
             << " available " << availableMemory
             << " total " << totalMemory << endl;
    }

    struct MemoryPoolsCollection
    {
        DeviceMemoryPool<uint> uintVertices;
        DeviceMemoryPool<float> floatVertices;
        DeviceMemoryPool<uint> uintEdges;
        DeviceMemoryPool<float> floatEdges;
    };

    static const uint kUintVerticesPoolsRequired = 8;
    static const uint kFloatVerticesPoolsRequired = 3;
    static const uint kUintEdgesPoolsRequired = 8;
    static const uint kFloatEdgesPoolsRequired = 4;

    void initalizeData(const Graph &graph, MemoryPoolsCollection &pools)
    {
        // Get memory for the internal variables
        verticesCount_ = static_cast<uint>(graph.vertices.size());
        edgesCount_ = static_cast<uint>(graph.edges.size());

        dVertices_ = pools.uintVertices.get();
        dEdges_ = pools.uintEdges.get();
        dWeights_ = pools.floatEdges.get();

        dOutputEdgesFlags_ = pools.uintEdges.get();

        // Copy graph to the device memory
        checkCudaErrors(hipMemcpy(dVertices_.get(), &(graph.vertices[0]), sizeof(uint) * verticesCount_, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dEdges_.get(), &(graph.edges[0]), sizeof(uint) * edgesCount_, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dWeights_.get(), &(graph.weights[0]), sizeof(float) * edgesCount_, hipMemcpyHostToDevice));

        thrust::fill(dOutputEdgesFlags_, dOutputEdgesFlags_ + edgesCount_, 0);
    }

    // static const uint kMaxThreadsPerBlock = 256;
    static const uint kMaxThreadsPerBlock = 1024;

    // Calculates grid parameters of the consecutive kernel calls
    // based on the number of elements in the array.
    void calculateThreadsDistribution(uint totalElements, uint &blocksCount, uint &threadsPerBlockCount)
    {
        if (totalElements > kMaxThreadsPerBlock)
        {
            blocksCount = (totalElements + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;

            threadsPerBlockCount = kMaxThreadsPerBlock;
        }
        else
        {
            blocksCount = 1;
            threadsPerBlockCount = totalElements;
        }
    }

    enum AlgorithmStatus
    {
        ALGORITHM_NOT_FINISHED,
        ALGORITHM_FINISHED
    };

    AlgorithmStatus invokeStep(MemoryPoolsCollection &pools, Pyramid &segmentations, int3 num_image_voxels_xyz)
    {
        // cout << "invokeStep()" << endl;
        uint blocksCount, threadsPerBlockCount;
        int x, y, z;
        x = num_image_voxels_xyz.x;
        y = num_image_voxels_xyz.y;
        z = num_image_voxels_xyz.z;

        calculateThreadsDistribution(edgesCount_, blocksCount, threadsPerBlockCount);
        dim3 gridDimsForEdges(blocksCount, 1, 1);
        dim3 blockDimsForEdges(threadsPerBlockCount, 1, 1);

        calculateThreadsDistribution(verticesCount_, blocksCount, threadsPerBlockCount);
        dim3 gridDimsForVertices(blocksCount, 1, 1);
        dim3 blockDimsForVertices(threadsPerBlockCount, 1, 1);

        thrust::device_ptr<uint> dEdgesFlags = pools.uintEdges.get();

        thrust::fill(dEdgesFlags, dEdgesFlags + edgesCount_, 0);

        // 1) Maring the MST edges: Finding minimum weighted edge

        // Mark the first edge for each vertex in "dEdgesFlags"
        markSegments<<<gridDimsForVertices, blockDimsForVertices, 0>>>(dVertices_.get(), dEdgesFlags.get(), verticesCount_);
        // hipDeviceSynchronize();
        getLastCudaError("markSegments launch failed.");

        // Now find minimum edges for each vertex.
        thrust::device_ptr<uint> dMinScannedEdges = pools.uintEdges.get();
        thrust::device_ptr<float> dMinScannedWeights = pools.floatEdges.get();

        thrust::inclusive_scan_by_key(dEdgesFlags,
                                      dEdgesFlags + edgesCount_,
                                      thrust::make_zip_iterator(thrust::make_tuple(dWeights_, dEdges_)),
                                      thrust::make_zip_iterator(thrust::make_tuple(dMinScannedWeights, dMinScannedEdges)),
                                      thrust::greater_equal<uint>(),
                                      thrust::minimum<thrust::tuple<float, uint>>());

        // To make things clear.
        // Let "dEdgesFlags" denote groups of edges that
        // correspond to the same vertices. Then the last edge of each group
        // (in "dMinScannedEdges" and "dMinScannedWeights") is now minimal.

        // 2) Marking the MST edges: Finding and removing cycles //

        // Calculate a successor vertex for each vertex. A successor of the
        // vertex v is a neighbouring vertex connected to v
        // by the minimal edge.
        thrust::device_ptr<uint> dSuccessors = pools.uintVertices.get();

        getSuccessors<<<gridDimsForVertices, blockDimsForVertices, 0>>>(dVertices_.get(), dMinScannedEdges.get(), dSuccessors.get(), verticesCount_, edgesCount_);
        // hipDeviceSynchronize();
        getLastCudaError("getSuccessors launch failed.");

        pools.uintEdges.put(dMinScannedEdges);
        pools.floatEdges.put(dMinScannedWeights);

        // Remove cyclic successor dependencies. Note that there can be only
        // two vertices in a cycle. See fig. 7 & fig. 8 in [1] for details.
        removeCycles<<<gridDimsForVertices, blockDimsForVertices, 0>>>(dSuccessors.get(), verticesCount_);
        // hipDeviceSynchronize();
        getLastCudaError("removeCycles launch failed.");

        // 3) Graph Construction: Merging vertices //

        // Build up an array of startpoints for edges. As already stated,
        // each group of edges denoted by "dEdgesFlags"
        // has the same startpoint.
        thrust::device_ptr<uint> dStartpoints = pools.uintEdges.get();
        thrust::inclusive_scan(dEdgesFlags, dEdgesFlags + edgesCount_, dStartpoints);
        addScalar<<<gridDimsForEdges, blockDimsForEdges, 0>>>(dStartpoints.get(), -1, edgesCount_);
        // hipDeviceSynchronize();
        getLastCudaError("addScalar launch failed.");

        // Shrink the chains of successors. New successors will eventually
        // represent superpixels of the new level.
        thrust::device_ptr<uint> dRepresentatives = pools.uintVertices.get();

        getRepresentatives<<<gridDimsForVertices, blockDimsForVertices, 0>>>(dSuccessors.get(), dRepresentatives.get(), verticesCount_);
        // hipDeviceSynchronize();
        getLastCudaError("getRepresentatives launch failed.");

        swap(dSuccessors, dRepresentatives);

        pools.uintVertices.put(dRepresentatives);

        // 4) Graph Construction: Assigned ids to superpixels //

        // Group vertices by successors' indices.
        thrust::device_ptr<uint> dClusteredVerticesIDs = pools.uintVertices.get();

        thrust::sequence(dClusteredVerticesIDs, dClusteredVerticesIDs + verticesCount_);

        thrust::sort(
            thrust::make_zip_iterator(
                thrust::make_tuple(thrust::device_ptr<uint>(dSuccessors),
                                   thrust::device_ptr<uint>(dClusteredVerticesIDs))),
            thrust::make_zip_iterator(
                thrust::make_tuple(thrust::device_ptr<uint>(dSuccessors + verticesCount_),
                                   thrust::device_ptr<uint>(dClusteredVerticesIDs + verticesCount_))));

        // Mark those groups.
        thrust::device_ptr<uint> dVerticesFlags_ = pools.uintVertices.get();

        thrust::fill(dVerticesFlags_, dVerticesFlags_ + verticesCount_, 0);

        thrust::adjacent_difference(dSuccessors, dSuccessors + verticesCount_, dVerticesFlags_, thrust::not_equal_to<uint>());

        hipMemset((void *)dVerticesFlags_.get(), 0, sizeof(uint));

        // Assign new indices to the successors (the indices of vertices
        // at the new level).
        thrust::device_ptr<uint> dNewVerticesIDs_ = pools.uintVertices.get();

        thrust::inclusive_scan(dVerticesFlags_, dVerticesFlags_ + verticesCount_, dNewVerticesIDs_);

        pools.uintVertices.put(dVerticesFlags_);

        // Now we can calculate number of resulting superpixels easily.
        uint newVerticesCount;
        hipMemcpy(&newVerticesCount, (dNewVerticesIDs_ + verticesCount_ - 1).get(), sizeof(uint), hipMemcpyDeviceToHost);
        ++newVerticesCount;

        // There are two special cases when we can stop our algorithm:
        // 1) number of vertices in the graph remained unchanged;
        // 2) only one vertex remains.
        if (newVerticesCount == verticesCount_)
        {
            cout << "No. of vertices remained unchanged, finishing" << endl;
            return ALGORITHM_FINISHED;
        }
        else if (newVerticesCount == 1)
        {
            thrust::device_ptr<uint> dDummyVerticesOffsets = pools.uintVertices.get();

            hipMemset((void *)dDummyVerticesOffsets.get(), 0, sizeof(uint));

            thrust::device_ptr<uint> dDummyVerticesIDs = pools.uintVertices.get();

            thrust::sequence(dDummyVerticesIDs, dDummyVerticesIDs + verticesCount_);

            segmentations.addLevel(1, verticesCount_, dDummyVerticesOffsets, dDummyVerticesIDs);

            // cout << "Only one vertex remains, finishing" << endl;
            return ALGORITHM_FINISHED;
        }

        // 5) Graph Construction: Removing and forming the new edge list //

        // Calculate how old vertices IDs map to new vertices IDs.
        thrust::device_ptr<uint> dVerticesMapping = pools.uintVertices.get();

        getVerticesMapping<<<gridDimsForVertices, blockDimsForVertices, 0>>>(dClusteredVerticesIDs.get(), dNewVerticesIDs_.get(), dVerticesMapping.get(), verticesCount_);
        // hipDeviceSynchronize();
        getLastCudaError("getVerticesMapping launch failed.");

        pools.uintVertices.put(dNewVerticesIDs_);
        pools.uintVertices.put(dClusteredVerticesIDs);
        pools.uintVertices.put(dSuccessors);

        // Invalidate self-loops in the reduced graph (the graph
        // produced by merging all old vertices that have
        // the same successor).
        invalidateLoops<<<gridDimsForEdges, blockDimsForEdges, 0>>>(dStartpoints.get(), dVerticesMapping.get(), dEdges_.get(), edgesCount_);
        // hipDeviceSynchronize();
        getLastCudaError("invalidateLoops launch failed.");

        // Calculate various information about the surviving
        // (new startpoints IDs and IDs of edges) and
        // non-surviving/contracted edges (their weights).
        thrust::device_ptr<uint> dNewStartpoints = pools.uintEdges.get();
        thrust::device_ptr<uint> dSurvivedEdgesIDs = pools.uintEdges.get();

        calculateEdgesInfo<<<gridDimsForEdges, blockDimsForEdges, 0>>>(dStartpoints.get(), dVerticesMapping.get(), dEdges_.get(), dWeights_.get(), dNewStartpoints.get(), dSurvivedEdgesIDs.get(), edgesCount_, newVerticesCount);
        // hipDeviceSynchronize();
        getLastCudaError("calculateEdgesInfo launch failed.");

        pools.uintEdges.put(dStartpoints);

        // Group that information by the new startpoints IDs.
        // Keep in mind that we want to build new (reduced) graph and apply
        // the step of the algorithm to that one. Hence we need to
        // preserve the structure of the original graph: neighbours and
        // weights should be grouped by vertex.
        thrust::sort(
            thrust::make_zip_iterator(
                thrust::make_tuple(dNewStartpoints,
                                   dSurvivedEdgesIDs)),
            thrust::make_zip_iterator(
                thrust::make_tuple(dNewStartpoints + edgesCount_,
                                   dSurvivedEdgesIDs + edgesCount_)));

        // Find the group of contracted edges.
        uint *invalidEdgesPtr =
            thrust::find_if(
                dNewStartpoints,
                dNewStartpoints + edgesCount_,
                IsGreaterEqualThan<uint>(newVerticesCount))
                .get();

        // Calculate how many edges there are in the reduced graph.
        uint validEdgesCount = static_cast<uint>(invalidEdgesPtr - dNewStartpoints.get());

        // Mark groups of edges corresponding to the same vertex in the
        // reduced graph.
        thrust::adjacent_difference(dNewStartpoints, dNewStartpoints + edgesCount_, dEdgesFlags, thrust::not_equal_to<uint>());

        hipMemset((void *)dEdgesFlags.get(), 0, sizeof(uint));
        hipMemset((void *)dEdgesFlags.get(), 1, 1);

        pools.uintEdges.put(dNewStartpoints);

        // 6) Graph Construction: Constructing the vertex list //

        // Now we are able to build the reduced graph. See "Graph"
        // class for the details on the graph's internal structure.

        // Calculate vertices' offsets for the reduced graph.
        thrust::copy_if(thrust::make_counting_iterator(0U), thrust::make_counting_iterator(validEdgesCount), dEdgesFlags, dVertices_, thrust::identity<uint>()).get();

        pools.uintEdges.put(dEdgesFlags);

        // Build up a neighbourhood for each vertex in the reduced graph
        // (this includes recalculating edges' weights).
        calculateThreadsDistribution(validEdgesCount, blocksCount, threadsPerBlockCount);
        dim3 newGridDimsForEdges(blocksCount, 1, 1);
        dim3 newBlockDimsForEdges(threadsPerBlockCount, 1, 1);

        thrust::device_ptr<uint> dNewEdges = pools.uintEdges.get();
        thrust::device_ptr<float> dNewWeights = pools.floatEdges.get();

        makeNewEdges<<<newGridDimsForEdges, newBlockDimsForEdges, 0>>>(dSurvivedEdgesIDs.get(), dVerticesMapping.get(), dEdges_.get(), dWeights_.get(), dNewEdges.get(), dNewWeights.get(), validEdgesCount);
        // hipDeviceSynchronize();
        getLastCudaError("makeNewEdges launch failed.");

        swap(dEdges_, dNewEdges);
        swap(dWeights_, dNewWeights);

        pools.uintEdges.put(dNewEdges);
        pools.floatEdges.put(dNewWeights);

        pools.uintEdges.put(dSurvivedEdgesIDs);

        // The graph's reconstruction is now finished.

        // Build new level of the segmentation tree. It is a trivial task
        // as we already have "dVerticesMapping" that contains all
        // sufficient information about the vertices' transformations.
        thrust::device_ptr<uint> dVerticesIDs = pools.uintVertices.get();
        thrust::device_ptr<uint> dNewVerticesOffsets = pools.uintVertices.get();

        thrust::sequence(dVerticesIDs, dVerticesIDs + verticesCount_);

        thrust::sort_by_key(dVerticesMapping, dVerticesMapping + verticesCount_, dVerticesIDs);

        thrust::unique_by_key_copy(dVerticesMapping, dVerticesMapping + verticesCount_, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), dNewVerticesOffsets);

        segmentations.addLevel(newVerticesCount, verticesCount_, dNewVerticesOffsets, dVerticesIDs);

        pools.uintVertices.put(dVerticesIDs);
        pools.uintVertices.put(dNewVerticesOffsets);
        pools.uintVertices.put(dVerticesMapping);

        // We can now safely set new counts for vertices and edges.
        verticesCount_ = newVerticesCount;
        edgesCount_ = validEdgesCount;

        return ALGORITHM_NOT_FINISHED;
    }

    uint verticesCount_;
    uint edgesCount_;

    thrust::device_ptr<uint> dVertices_;
    thrust::device_ptr<uint> dEdges_;
    thrust::device_ptr<float> dWeights_;

    thrust::device_ptr<uint> dOutputEdgesFlags_;
};

// inline float distance(const uchar3 &first, const uchar3 &second)
inline float distance(const float &first, const float &second)
{
    return abs(first - second);
}

inline int getSingleIndex(int3 num_image_voxels_xyz, int ix, int iy, int iz)
{
    int index = INDEX3D(iz, ix, iy, num_image_voxels_xyz.z, num_image_voxels_xyz.x);
    return index;
}

// Builds a net-graph for the image with 4-connected pixels.
// Builds a net-graph for the image with 4-connected pixels.
// void buildGraph(const vector<uchar3> &image, uint width, uint height, Graph &graph)
void buildGraph(float *image, int3 num_image_voxels_xyz, Graph &graph)
{
    uint connected_6[6][3] = {{0, 0, 1},
                              {0, 1, 0},
                              {1, 0, 0},
                              {-1, 0, 0},
                              {0, -1, 0},
                              {0, 0, -1}};

    uint connectedArrayLength = sizeof(connected_6) / sizeof(connected_6[0]);
    cout << "connectedArrayLength: " << connectedArrayLength << endl;

    uint total_x, total_y, total_z, totalNodes, totalEdges;
    total_x = num_image_voxels_xyz.x;
    total_y = num_image_voxels_xyz.y;
    total_z = num_image_voxels_xyz.z;
    totalNodes = size(num_image_voxels_xyz);
    totalEdges = (6 * total_x * total_y * total_z) - (2 * total_x * total_z) - (2 * total_y * total_z) - (2 * total_x * total_y); // formula for edges in 6-con 3D lattice by x,y,z

    graph.vertices.resize(totalNodes);
    graph.edges.reserve(totalEdges);
    graph.weights.reserve(totalEdges);

    uint edgesProcessed = 0;

    for (uint y = 0; y < total_y; ++y)
    {
        for (uint x = 0; x < total_x; ++x)
        {
            for (uint z = 0; z < total_z; ++z)
            {

                uint nodeIndex = getSingleIndex(num_image_voxels_xyz, x, y, z);

                const float &centerPixel = image[nodeIndex];

                graph.vertices[nodeIndex] = edgesProcessed;

                // 6 connected (faces)
                for (int i = 0; i < connectedArrayLength; i++) // double edged edges
                {
                    int x_component = x + connected_6[i][0];
                    int y_component = y + connected_6[i][1];
                    int z_component = z + connected_6[i][2];

                    if (x_component >= 0 && y_component >= 0 && z_component >= 0 && x_component < num_image_voxels_xyz.x && y_component < num_image_voxels_xyz.y && z_component < num_image_voxels_xyz.z) // if boundary pixels, avoids get seg fault
                    {
                        uint neighbourNode = getSingleIndex(num_image_voxels_xyz, x_component, y_component, z_component);

                        const float &neighbourPixel = image[neighbourNode];

                        graph.edges.push_back(neighbourNode);
                        graph.weights.push_back(distance(centerPixel, neighbourPixel));

                        ++edgesProcessed;
                    }
                }
                // }
            }
        }
    }

    graph.vertices.shrink_to_fit();
    graph.edges.shrink_to_fit();
    graph.weights.shrink_to_fit();
}